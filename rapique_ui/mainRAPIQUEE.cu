#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <string>
#include <filesystem>
#include <opencv2/opencv.hpp>
#include <opencv2/core/ocl.hpp>
#include <chrono>
#include "Timer.h"
#include "Globals.h"
#include "Logger.h"
#include <hip/hip_runtime.h>

__global__ void copyFeatsToMat(float* featsFramesData, float* featsFramesMatData, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row = idx / cols;
    int col = idx % cols;
    if (row < rows && col < cols) {
        featsFramesMatData[row * cols + col] = featsFramesData[row * cols + col];
    }
}

__global__ void averageColumns(float* input, float* output, int rows, int cols) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (col < cols) {
        float sum = 0;
        for (int row = 0; row < rows; ++row) {
            sum += input[row * cols + col];
        }
        output[col] = sum / rows;
    }
}

std::vector<float> flatten(const std::vector<std::vector<float>>& vec) {
    std::vector<float> flat;
    for (const auto& subVec : vec) {
        flat.insert(flat.end(), subVec.begin(), subVec.end());
    }
    return flat;
}

using namespace std::chrono;

std::vector<std::vector<float>> calc_RAPIQUE_features(const std::string& yuv_name, int width, int height,
    int framerate, float minside, const std::string& net,
    const std::string& layer, int log_level);

struct DataRow {
    long long flickr_id;
    float mos;
    int width;
    int height;
    std::string pixfmt;
    float framerate;
    int nb_frames;
    int bitdepth;
    int bitrate;
    DataRow(const std::string& line) {
        std::istringstream iss(line);
        std::string token;
        std::getline(iss, token, ','); flickr_id = std::stoll(token);
        std::getline(iss, token, ','); mos = std::stod(token);
        std::getline(iss, token, ','); width = std::stoi(token);
        std::getline(iss, token, ','); height = std::stoi(token);
        std::getline(iss, token, ','); pixfmt = token;
        std::getline(iss, token, ','); framerate = std::stod(token);
        std::getline(iss, token, ','); nb_frames = std::stoi(token);
        std::getline(iss, token, ','); bitdepth = std::stoi(token);
        std::getline(iss, token, ','); bitrate = std::stoi(token);
    }
};

void writeXML(const std::string& filename, const std::vector<std::vector<float>>& feats_frames) {
    std::ofstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Error opening the file for writing.\n";
        return;
    }

    file << "<?xml version=\"1.0\"?>" << std::endl
        << "<opencv_storage>" << std::endl
        << "<FeatFrames type_id=\"opencv-matrix\">" << std::endl
        << "<rows>" << feats_frames.size() << "</rows>" << std::endl
        << "<cols>" << feats_frames[0].size() << "</cols>" << std::endl
        << "<dt>" << "d" << "</dt>" << std::endl
        << "<data>" << std::endl;
    file << std::fixed << std::setprecision(17);
    for (const auto& row : feats_frames) {
        for (const auto& elem : row) {
            file << elem << " ";
        }
        file << std::endl;
    }
    file << "</data>" << std::endl
        << "</FeatFrames>" << std::endl
        << "</opencv_storage>" << std::endl;
    file.close();
}

int main_RAPIQUEE() {
    logger.startTimer("RAPIQUE-Main", "demo_compute_RAPIQUE_feats");
    Timer timer;
    if (cv::ocl::haveOpenCL()) {
        cv::ocl::setUseOpenCL(true);
        std::cout << "OpenCL support detected. Using GPU..." << std::endl;
    }
    else {
        std::cout << "OpenCL support not detected. Using CPU..." << std::endl;
    }

    std::cout << "Current Working Directory: " << std::filesystem::current_path() << std::endl;


    const std::string path_separator = "\\";
    const std::string dataBase_path = "dataBase";
    const std::string featureFrames_output_path = "output";

    std::string filelist_csv = "mos_file.csv";

    std::vector<DataRow> filelist;

    std::ifstream inFile(filelist_csv);
    if (inFile.is_open()) {
        std::string line;
        std::getline(inFile, line); // Skip the header line.
        while (std::getline(inFile, line)) {
            try {
                filelist.push_back(DataRow(line));
            }
            catch (const std::exception& e) {
                std::cerr << "Error processing line: " << line << ". Cause: " << e.what() << std::endl;
            }
        }
        inFile.close();
    }
    else {
        std::cerr << "Unable to open file " << filelist_csv << std::endl;
    }

    if (!std::filesystem::exists(featureFrames_output_path)) {
        std::filesystem::create_directory(featureFrames_output_path);
    }

    std::vector<std::vector<float>> feats_mat(filelist.size());

    // Using a loop index to safely access and modify feats_mat
    for (size_t index = 0; index < filelist.size(); ++index) {
        const auto& entry = filelist[index];
        timer.start();
        std::cout << "\n\n ----------->Computing features for " << entry.flickr_id << " sequence\n";
        std::string video_name = dataBase_path + path_separator + std::to_string(entry.flickr_id) + ".mp4";
        std::string yuv_name = featureFrames_output_path + path_separator + std::to_string(entry.flickr_id) + ".yuv";
        std::string cmd = "ffmpeg -loglevel error -y -i " + video_name + " -pix_fmt yuv420p -vsync 0 " + yuv_name;
        system(cmd.c_str());

        std::vector<std::vector<float>> feats_frames = calc_RAPIQUE_features(yuv_name, entry.width, entry.height,
            std::round(entry.framerate), 512.0f, "resnet50", "avg_pool", 0);
        std::vector<float> flat_feats_frames = flatten(feats_frames);
        int numElements = flat_feats_frames.size();
        float* d_featsFrames;
        float* d_featsFramesMat;
        hipError_t allocStatus = hipMalloc(&d_featsFrames, numElements * sizeof(float));
        if (allocStatus != hipSuccess) {
            std::cerr << "CUDA error: Failed to allocate d_featsFrames: " << hipGetErrorString(allocStatus) << std::endl;
            continue;
        }
        hipMemcpy(d_featsFrames, flat_feats_frames.data(), numElements * sizeof(float), hipMemcpyHostToDevice);
        allocStatus = hipMalloc(&d_featsFramesMat, numElements * sizeof(float));
        if (allocStatus != hipSuccess) {
            std::cerr << "CUDA error: Failed to allocate d_featsFramesMat: " << hipGetErrorString(allocStatus) << std::endl;
            hipFree(d_featsFrames);
            continue;
        }

        dim3 blockSize(256);
        dim3 gridSize((numElements + blockSize.x - 1) / blockSize.x);
        copyFeatsToMat << <gridSize, blockSize >> > (d_featsFrames, d_featsFramesMat, feats_frames.size(), feats_frames[0].size());
        hipDeviceSynchronize();

        float* d_meanMat;
        hipMalloc(&d_meanMat, feats_frames[0].size() * sizeof(float));
        averageColumns << <gridSize, blockSize >> > (d_featsFramesMat, d_meanMat, feats_frames.size(), feats_frames[0].size());
        hipDeviceSynchronize();

        std::vector<float> meanMat(feats_frames[0].size());
        hipMemcpy(meanMat.data(), d_meanMat, feats_frames[0].size() * sizeof(float), hipMemcpyDeviceToHost);
        feats_mat[index] = meanMat;

        hipFree(d_featsFrames);
        hipFree(d_featsFramesMat);
        hipFree(d_meanMat);
        std::remove(yuv_name.c_str());

            writeXML(featureFrames_output_path + path_separator + "RAPIQUE_feats.xml", feats_mat);
            std::cout << "XML file saved." << std::endl;
        
        float time = timer.elapsed();
        std::cout << "The code was executed in: " << time << " seconds." << std::endl;
    }

    logger.stopTimer("RAPIQUE-Main", "demo_compute_RAPIQUE_feats");
    logger.writeXML();
    return 0;
}
