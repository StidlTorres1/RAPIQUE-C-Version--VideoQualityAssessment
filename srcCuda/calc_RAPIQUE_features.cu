#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <cmath> 
#include <fstream>
#include <opencv2/opencv.hpp>
#include <opencv2/core/ocl.hpp>
#include <cstdio>
#include <algorithm>
#include <execution>
#include <mutex>
#include <torch/script.h> 
#include <iostream>
#include <memory>
#include <iomanip>

#include "ImageReaderFactory.h"
#include "Logger.h"
#include "Globals.h"

#include <opencv2/core/core.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudafilters.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/cudawarping.hpp>
#include <opencv2/cudaimgproc.hpp> 



using namespace std;
std::mutex mtx;

torch::jit::Module loadModel(const std::string& modelPath) {
	torch::jit::Module module;
	torch::Device device(torch::kCUDA);
	try {
		std::cout << "Loading model\n";
		module = torch::jit::load(modelPath);
		std::cout << "Model loaded\n";
	}
	catch (const c10::Error& e) {
		std::cerr << "error loading the model\n";
		exit(-1); 
	}
	return module;
}

std::vector<float> loadWfun() {
	return std::vector<float>{
		1, 1, 1, 1, -1, -1, -1, -1,
			1, 1, -1, -1, 1, 1, -1, -1,
			1, 1, -1, -1, -1, -1, 1, 1,
			1, -1, 1, -1, 1, -1, 1, -1,
			1, -1, 1, -1, -1, 1, -1, 1,
			1, -1, -1, 1, 1, -1, -1, 1,
			
			1, -1, -1, 1, -1, 1, 1, -1
	};
}

vector<float> rapique_basic_extractor(const cv::Mat& src);
void process_channel(int ch, int kscale, const std::vector<cv::Mat>& dpt_filt_frames, float ratio, std::vector<std::vector<float>>& feats_tmp_wpt_global) {
	std::vector<std::vector<float>> feats_tmp_wpt_local;

	cv::Mat feat_map;
	if (ratio < 1) {
		cv::resize(dpt_filt_frames[ch], feat_map, cv::Size(), ratio, ratio, cv::INTER_AREA);
	}
	else {
		feat_map = dpt_filt_frames[ch];
	}

	for (int scale = 1; scale <= kscale; ++scale) {
		cv::Mat y_scale;
		if (scale > 1) {
			float factor = std::pow(2.0, -(scale - 1));
			cv::resize(feat_map, y_scale, cv::Size(), factor, factor, cv::INTER_AREA);
		}
		else {
			y_scale = feat_map;
		}

		auto features = rapique_basic_extractor(y_scale);
		feats_tmp_wpt_local.push_back(features);
	}

	std::lock_guard<std::mutex> lock(mtx);
	feats_tmp_wpt_global.insert(feats_tmp_wpt_global.end(), feats_tmp_wpt_local.begin(), feats_tmp_wpt_local.end());
}
vector<float> RAPIQUE_spatial_features(const cv::Mat& RGB);
vector<vector<float>> calc_RAPIQUE_features(const std::string& yuv_name, int width, int height,
    int framerate, float minside, const string& net, const string& layer, int log_level) {
    logger.startTimer(video_name_global, "calc_RAPIQUE_features"); // entry is name of the video file
    vector<vector<float>> feats_frames;

    try{
        bool modelLoaded = false;
        torch::jit::Module module;
        string modelPath = "../pytorch/traced_resnet50_avg_pool.pt";

        // Replaced direct YUV file handling with ImageReader abstract class
        auto imageReader = ImageReaderFactory::createImageReader("YUV");
        if (!imageReader) {
            cerr << "Error creating ImageReader instance.\n";
            return feats_frames;
        }

        // Use ImageReader to get the number of frames instead of manual file check
        ifstream test_file(yuv_name, ios::binary | ios::ate);
        if (!test_file.is_open()) {
            cerr << "Test YUV file not found.\n";
            return feats_frames;
        }
        streamsize file_length = test_file.tellg();
        test_file.close(); // Close the file as it's no longer needed directly

        int nb_frames = static_cast<int>(floor(file_length / (width * height * 1.5)));
        int half_framerate = framerate / 2;
        int third_framerate = framerate / 3;

        feats_frames.reserve(nb_frames * 2);

        vector<tuple<cv::Mat, cv::Mat, cv::Mat, int>> frame_triplets;
        frame_triplets.reserve(nb_frames);
        for (int fr = half_framerate; fr < nb_frames - 2; fr += framerate) {
            int calculatedFrame = max(1, fr - third_framerate);
            int frameIndex = min(nb_frames - 2, fr + third_framerate);

            frame_triplets.emplace_back(imageReader->readImage(yuv_name, width, height, fr),
                imageReader->readImage(yuv_name, width, height, calculatedFrame),
                imageReader->readImage(yuv_name, width, height, frameIndex), fr);
        }

        mutex mtx;
        for_each(std::execution::par, frame_triplets.begin(), frame_triplets.end(),
            [&]( tuple<cv::Mat, cv::Mat, cv::Mat, int>& frames) {
                 auto& [this_YUV_frame, prev_YUV_frame, next_YUV_frame, fr] = frames;

                // Check for empty frames is retained from your original logic
                if (this_YUV_frame.empty() || prev_YUV_frame.empty() || next_YUV_frame.empty()) {
                    cerr << "Error: One or more YUV frames are empty.\n";
                    return;
                }

                cv::Mat this_rgb, prev_rgb, next_rgb;
                cv::cvtColor(this_YUV_frame, this_rgb, cv::COLOR_YUV2BGR);
                cv::cvtColor(prev_YUV_frame, prev_rgb, cv::COLOR_YUV2BGR);
                cv::cvtColor(next_YUV_frame, next_rgb, cv::COLOR_YUV2BGR);

                this_YUV_frame.release();
                prev_YUV_frame.release();
                next_YUV_frame.release();

                // Resizing logic retains your original approach to adjusting frame size
                float sside = min(this_rgb.rows, this_rgb.cols);
                float ratio = minside / sside;
                if (ratio < 1) {
                    cv::resize(prev_rgb, prev_rgb, cv::Size(), ratio, ratio, cv::INTER_CUBIC);
                    cv::resize(next_rgb, next_rgb, cv::Size(), ratio, ratio, cv::INTER_CUBIC);
                }

                cv::cuda::Stream stream;

                vector<float> feats_per_frame;
                vector<float> prev_feats_spt = RAPIQUE_spatial_features(prev_rgb);
                vector<float> next_feats_spt = RAPIQUE_spatial_features(next_rgb);

                auto n_features = prev_feats_spt.size();
                vector<float> feats_spt_mean(n_features);
                vector<float> feats_spt_diff(n_features);

                transform(std::execution::par, prev_feats_spt.begin(), prev_feats_spt.end(), next_feats_spt.begin(),
                    feats_spt_mean.begin(), [](float a, float b) { return (a + b) / 2.0; });

                transform(std::execution::par, prev_feats_spt.begin(), prev_feats_spt.end(), next_feats_spt.begin(),
                    feats_spt_diff.begin(), [](float a, float b) { return abs(a - b); });

                lock_guard<mutex> guard(mtx);
                feats_per_frame.insert(feats_per_frame.end(), feats_spt_mean.begin(), feats_spt_mean.end());
                feats_per_frame.insert(feats_per_frame.end(), feats_spt_diff.begin(), feats_spt_diff.end());
                logger.startTimer(video_name_global, "calc_RAPIQUE_features/deep_learning"); // calc_RAPIQUE_features/deep_learning
                if (!modelLoaded) {
                    module = loadModel(modelPath);
                    modelLoaded = true;
                }

                cv::cuda::GpuMat d_image;
                d_image.upload(this_rgb);  // Upload to GPU memory

                // Resize image
                cv::cuda::resize(d_image, d_image, cv::Size(224, 224));

                // Convert color
                cv::cuda::cvtColor(d_image, d_image, cv::COLOR_BGR2RGB);

                // Convert to float and scale
                d_image.convertTo(d_image, CV_32F, 1.0 / 255);

                // Download from GPU to CPU
                cv::Mat image;
                d_image.download(image);  // Download to CPU memory
                d_image.release();

                // Continue with tensor operations on CPU
                auto img_tensor = torch::from_blob(image.data, { 1, 224, 224, 3 }, torch::kF32);
                img_tensor = img_tensor.permute({ 0, 3, 1, 2 });

                torch::NoGradGuard no_grad;
                auto output = module.forward({ img_tensor }).toTensor();

                output = output.squeeze();
                auto flattened_output = output.mean({ 1, 2 });
                auto feats = flattened_output.accessor<float, 1>();

                for (int i = 0; i < feats.size(0); ++i) {
                    feats_per_frame.push_back(static_cast<float>(feats[i]));
                }
                logger.stopTimer(video_name_global, "calc_RAPIQUE_features/deep_learning");// calc_RAPIQUE_features/deep_learning end
                logger.startTimer(video_name_global, "calc_RAPIQUE_features/temporal NSS");// calc_RAPIQUE_features/temporal NSS init
                std::vector<float> wfun = loadWfun();
                const int numRows = 7;

                int depth = wfun.size() / numRows;
                std::vector<cv::Mat> frames_wpt(depth, cv::Mat::zeros(prev_rgb.rows, prev_rgb.cols, CV_32FC1));

                int fr_idx_start = std::max(1, fr - static_cast<int>(std::floor(depth / 2.0)));
                int fr_idx_end = std::min(nb_frames - 3, fr_idx_start + depth - 1);

                int fr_wpt_cnt = 0;
                for (int fr_wpt = fr_idx_start; fr_wpt <= fr_idx_end; ++fr_wpt, ++fr_wpt_cnt) {
                    cv::Mat YUV_tmp = imageReader->readImage(yuv_name, width, height, fr_wpt);
                    cv::Mat processedFrame;
                    if (ratio < 1) {
                        cv::resize(YUV_tmp, processedFrame, cv::Size(), ratio, ratio);
                    }
                    else {
                        processedFrame = YUV_tmp;
                    }

                    if (fr_wpt_cnt < frames_wpt.size()) {
                        frames_wpt[fr_wpt_cnt] = processedFrame;
                    }
                }

                std::vector<std::vector<float>> wfunM = {
                    {1, 1, 1, 1, -1, -1, -1, -1},
                    {1, 1, -1, -1, 1, 1, -1, -1},
                    {1, 1, -1, -1, -1, -1, 1, 1},
                    {1, -1, 1, -1, 1, -1, 1, -1},
                    {1, -1, 1, -1, -1, 1, -1, 1},
                    {1, -1, -1, 1, 1, -1, -1, 1},
                    {1, -1, -1, 1, -1, 1, 1, -1}
                };
                std::vector<cv::Mat> processed_frames_wpt(frames_wpt.size());
                std::vector<cv::Mat> dpt_filt_frames(wfunM.size());

                for (size_t idx = 0; idx < frames_wpt.size(); ++idx) {
                    if (frames_wpt[idx].channels() > 1) {
                        cv::cvtColor(frames_wpt[idx], processed_frames_wpt[idx], cv::COLOR_BGR2GRAY);
                        processed_frames_wpt[idx].convertTo(processed_frames_wpt[idx], CV_32FC1);
                    }
                    else {
                        frames_wpt[idx].convertTo(processed_frames_wpt[idx], CV_32FC1);
                    }
                }

                cv::parallel_for_(cv::Range(0, wfunM.size()), [&](const cv::Range& range) {
                    for (int freq = range.start; freq < range.end; ++freq) {
                        cv::Mat sum_frame = cv::Mat::zeros(prev_rgb.rows, prev_rgb.cols, CV_32FC1);

                        for (size_t idx = 0; idx < processed_frames_wpt.size(); ++idx) {
                            sum_frame += processed_frames_wpt[idx] * wfun[freq];
                        }

                        dpt_filt_frames[freq] = sum_frame;
                    }
                    });

                int kscale = 2;
                vector<vector<float>> feats_tmp_wpt_global;

                cv::parallel_for_(cv::Range(0, dpt_filt_frames.size()), [&](const cv::Range& range) {
                    for (int ch = range.start; ch < range.end; ++ch) {
                        process_channel(ch, kscale, dpt_filt_frames, ratio, feats_tmp_wpt_global);
                    }
                    });

                vector<float> merged_feats;
                for (const auto& tmp_feats : feats_tmp_wpt_global) {
                    merged_feats.insert(merged_feats.end(), tmp_feats.begin(), tmp_feats.end());
                }

                feats_per_frame.insert(feats_per_frame.end(), merged_feats.begin(), merged_feats.end());
                feats_frames.push_back(feats_per_frame);

                stream.waitForCompletion();

                // Explicitly release GPU mats after processing
                this_rgb.release();
                prev_rgb.release();
                next_rgb.release();


            });
            logger.stopTimer(video_name_global, "calc_RAPIQUE_features/temporal NSS"); // calc_RAPIQUE_features/temporal NSS end

            logger.stopTimer(video_name_global, "calc_RAPIQUE_features");
            return feats_frames;
    }
    catch (const std::exception& e) {
        std::cerr << "An error occurred during feature calculation: " << e.what() << '\n';
        // Potentially rethrow or handle more specific exceptions here
        logger.stopTimer(video_name_global, "calc_RAPIQUE_features");
        throw;  // Rethrow the exception if you need to propagate it
    }
}
//Documentation
// Extracting and processing features from YUV video frames for video quality assessment. It uses OpenCV for image processing, along with other standard libraries for file and stream operations. Line by line:
// 1-8. Include statements:
// �	These lines include necessary headers for vector operations, string manipulation, mathematical functions, file streaming, OpenCV functionalities, OpenCL interface for GPU optimizations, C standard I/O, algorithm functions, execution policies for parallel algorithms, and mutex for thread safety.
// 9-11. Function declarations:
// �	Declares three functions YUVread, RAPIQUE_spatial_features, and calc_RAPIQUE_features that are defined elsewhere or later in the code.
// 12-76. Function calc_RAPIQUE_features:
// �	This function calculates RAPIQUE (Rapid and Accurate Image Quality Evaluator) features from a given YUV file.
// 13-17. File handling and initial checks:
// �	Opens a YUV file and checks if it's open. If not, it outputs an error message and returns an empty feature vector.
// 18-22. Frame number calculation:
// �	Calculates the number of frames in the YUV file based on its size and the dimensions of each frame.
// 23-27. Frame rate processing:
// �	Computes half_framerate and third_framerate for later use in determining which frames to process.
// 28-37. Frame triplet preparation:
// �	Reserves space for frame triplets and populates them by reading specific frames from the YUV file. It uses the YUVread function and adjusts frame indices based on the frame rate.
// 38.	Mutex declaration:
// �	Declares a mutex for thread safety during parallel processing.
// 39-75. Parallel processing of frames:
// �	Processes the frame triplets in parallel using for_each with execution::par policy.
// �	Converts YUV frames to RGB.
// �	Resizes the frames if necessary based on the minimum side length (minside) and aspect ratio.
// �	Extracts spatial features from previous and next frames using RAPIQUE_spatial_features.
// �	Calculates the mean and difference of features between the previous and next frames.
// �	Uses a mutex to safely add these features to the overall feature vector feats_frames.
// 77-83. XML file output:
// �	Writes the features to an XML file named "feat_frames.xml".
// �	Outputs a success message upon saving the file.
// 84.	Return statement:
// �	Returns the calculated features.
// This function is a comprehensive implementation for feature extraction from video frames, tailored for video quality assessment. It leverages parallel processing to efficiently handle multiple frames and computes spatial features for pairs of frames, which are likely used to assess temporal changes and overall video quality. The use of mutexes ensures thread safety during parallel execution. The features are then saved in an XML format for further use or analysis.
