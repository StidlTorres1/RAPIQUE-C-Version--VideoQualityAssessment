#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <string>
#include <filesystem>
#include <opencv2/opencv.hpp>
#include <opencv2/core/ocl.hpp>
#include <chrono>
#include "Timer.h" // Include the Timer class definition
#include "Globals.h" // Include the global variables definition
#include "Logger.h" // Include the logger functions definition
#include <vector>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>


__global__ void copyFeatsToMat(float* featsFramesData, float* featsFramesMatData, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row = idx / cols;
    int col = idx % cols;
    if (row < rows && col < cols) {
        featsFramesMatData[row * cols + col] = featsFramesData[row * cols + col];
    }
}

__global__ void averageColumns(float* input, float* output, int rows, int cols) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (col < cols) {
        float sum = 0;
        for (int row = 0; row < rows; ++row) {
            sum += input[row * cols + col];
        }
        output[col] = sum / rows;
    }
}

// Manually flatten the nested vector
std::vector<float> flatten(const std::vector<std::vector<float>>& vec) {
    std::vector<float> flat;
    for (const auto& subVec : vec) {
        flat.insert(flat.end(), subVec.begin(), subVec.end());
    }
    return flat;
}


using namespace std::chrono;

// Include libraries for file handling, image processing, and measuring execution time.

// Declare a function to calculate the RAPIQUE features from a video file.
std::vector<std::vector<float>> calc_RAPIQUE_features(const std::string& yuv_name, int width, int height,
    int framerate, float minside, const std::string& net,
    const std::string& layer, int log_level);

// Define a structure to hold video data information.
struct DataRow {
    long long flickr_id;
    float mos;
    int width;
    int height;
    std::string pixfmt;
    float framerate;
    int nb_frames;
    int bitdepth;
    int bitrate;

    // Constructor to parse a CSV line into a DataRow object.
    DataRow(const std::string& line) {
        std::istringstream iss(line);
        std::string token;
        std::getline(iss, token, ','); flickr_id = std::stoll(token);
        std::getline(iss, token, ','); mos = std::stod(token);
        std::getline(iss, token, ','); width = std::stoi(token);
        std::getline(iss, token, ','); height = std::stoi(token);
        std::getline(iss, token, ','); pixfmt = token;
        std::getline(iss, token, ','); framerate = std::stod(token);
        std::getline(iss, token, ','); nb_frames = std::stoi(token);
        std::getline(iss, token, ','); bitdepth = std::stoi(token);
        std::getline(iss, token, ','); bitrate = std::stoi(token);
    }
};

// Write the features to an XML file.
void writeXML(const std::string& filename, const std::vector<std::vector<float>>& feats_frames) {
    std::ofstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Error opening the file for writing.\n";
        return;
    }

    file << "<?xml version=\"1.0\"?>" << std::endl
        << "<opencv_storage>" << std::endl
        << "<FeatFrames type_id=\"opencv-matrix\">" << std::endl
        << "<rows>" << feats_frames.size() << "</rows>" << std::endl
        << "<cols>" << feats_frames[0].size() << "</cols>" << std::endl
        << "<dt>" << "d" << "</dt>" << std::endl
        << "<data>" << std::endl;
    file << std::fixed << std::setprecision(17);
    for (const auto& row : feats_frames) {
        for (const auto& elem : row) {
            file << elem << " ";
        }
        file << std::endl;
    }
    file << "</data>" << std::endl
        << "</FeatFrames>" << std::endl
        << "</opencv_storage>" << std::endl;
    file.close();
}

// Main function to process videos and compute features.
int main_RAPIQUEE() {
    logger.startTimer("RAPIQUE-Main", "demo_compute_RAPIQUE_feats");//  demo_compute_RAPIQUE_feats starts
    Timer timer;
    // Enable GPU support if available.
    if (cv::ocl::haveOpenCL()) {
        cv::ocl::setUseOpenCL(true);
        std::cout << "OpenCL support detected. Using GPU..." << std::endl;
    }
    else {
        std::cout << "OpenCL support not detected. Using CPU..." << std::endl;
    }

    // Configure paths based on operating system.
    const std::string path_separator =
#ifdef _WIN32
        "\\";
#else
        "/";
#endif

    // Define basic parameters for the process.
    const std::string algo_name = "RAPIQUE";
    const std::string data_name = "KONVID_1K";
    const bool write_file = true;
    std::filesystem::path currentPath = std::filesystem::current_path();
    std::filesystem::path desiredPath = currentPath.parent_path();
    std::string root_path = desiredPath.string() + path_separator + "dataBase" + path_separator;
    std::string root_path_data = root_path + data_name + path_separator;
    std::string data_path = root_path_data;
    std::string filelist_csv = root_path + "mos_files" + path_separator + "KONVID_1K_1test_metadata.csv";

    // Read video file metadata.
    std::vector<DataRow> filelist;
    std::ifstream inFile(filelist_csv);
    if (inFile.is_open()) {
        std::string line;
        std::getline(inFile, line); // Skip the header line.
        while (std::getline(inFile, line)) {
            try {
                filelist.push_back(DataRow(line));
            }
            catch (const std::exception& e) {
                std::cerr << "Error processing line: " << line << ". Cause: " << e.what() << std::endl;
            }
        }
        inFile.close();
    }
    else {
        std::cerr << "Unable to open file " << filelist_csv << std::endl;
    }

    // Create directories for output if they do not exist.
    std::string out_path = root_path + "feat_files";
    std::string out_path_temp = root_path + "tmp";
    if (!std::filesystem::exists(out_path)) {
        std::filesystem::create_directory(out_path);
    }
    if (!std::filesystem::exists(out_path_temp)) {
        std::filesystem::create_directory(out_path_temp);
    }

    std::vector<std::vector<float>> feats_mat(filelist.size());
    int h = 0;
    for (const auto& entry : filelist) {
        timer.start();
        std::cout << "\n\n ----------->Computing features for " << entry.flickr_id << " sequence\n";
        std::string video_name = data_path + path_separator + std::to_string(entry.flickr_id) + ".mp4";
        std::string yuv_name = out_path_temp + path_separator + std::to_string(entry.flickr_id) + ".yuv";
        std::string cmd = "ffmpeg -loglevel error -y -i " + video_name + " -pix_fmt yuv420p -vsync 0 " + yuv_name;
        system(cmd.c_str());

        std::vector<std::vector<float>> feats_frames = calc_RAPIQUE_features(yuv_name, entry.width, entry.height,
            std::round(entry.framerate), 512.0f, "resnet50", "avg_pool", 0);

        std::vector<float> flat_feats_frames = flatten(feats_frames);
        int numElements = flat_feats_frames.size();
        float* d_featsFrames;
        float* d_featsFramesMat;

        hipError_t allocStatus;

        // Allocate memory for d_featsFrames
        allocStatus = hipMalloc(&d_featsFrames, numElements * sizeof(float));
        if (allocStatus != hipSuccess) {
            std::cerr << "CUDA error: Failed to allocate d_featsFrames: " << hipGetErrorString(allocStatus) << std::endl;
            continue; // Skip to the next iteration of the loop
        }

        // Copy host memory to device
        hipMemcpy(d_featsFrames, flat_feats_frames.data(), numElements * sizeof(float), hipMemcpyHostToDevice);

        // Allocate memory for d_featsFramesMat
        allocStatus = hipMalloc(&d_featsFramesMat, numElements * sizeof(float)); // Correcting the missing allocation
        if (allocStatus != hipSuccess) {
            std::cerr << "CUDA error: Failed to allocate d_featsFramesMat: " << hipGetErrorString(allocStatus) << std::endl;
            hipFree(d_featsFrames); // Free previously allocated memory before continuing
            continue; // Skip to the next iteration of the loop
        }

        dim3 blockSize(256);
        dim3 gridSize((numElements + blockSize.x - 1) / blockSize.x);

        // Now that d_featsFramesMat is properly allocated, call the kernel
        copyFeatsToMat << <gridSize, blockSize >> > (d_featsFrames, d_featsFramesMat, feats_frames.size(), feats_frames[0].size());
        hipDeviceSynchronize();

        float* d_meanMat;
        hipMalloc(&d_meanMat, feats_frames[0].size() * sizeof(float));
        averageColumns << <gridSize, blockSize >> > (d_featsFramesMat, d_meanMat, feats_frames.size(), feats_frames[0].size());
        hipDeviceSynchronize();

        std::vector<float> meanMat(feats_frames[0].size());
        hipMemcpy(meanMat.data(), d_meanMat, feats_frames[0].size() * sizeof(float), hipMemcpyDeviceToHost);

        if (h < feats_mat.size()) {
            feats_mat[h].resize(meanMat.size());
            for (int col = 0; col < meanMat.size(); ++col) {
                feats_mat[h][col] = meanMat[col];
            }
        }

        hipFree(d_featsFrames);
        hipFree(d_featsFramesMat);
        hipFree(d_meanMat);

        std::remove(yuv_name.c_str());
        if (write_file) {
            writeXML(out_path + path_separator + data_name + "_" + algo_name + "_feats.xml", feats_mat);
            std::cout << "XML file saved." << std::endl;
            std::cout << "Features processed: " << feats_frames.size() << std::endl;
            std::cout << "Features dimensions: " << feats_frames[0].size() << std::endl;
        }
        float time = timer.elapsed();
        std::cout << "The code was executed in: " << time << " seconds." << std::endl;
        h++;
    }


    logger.stopTimer("RAPIQUE-Main", "demo_compute_RAPIQUE_feats");//  demo_compute_RAPIQUE_feats end
    logger.writeXML();
    return 0;
}