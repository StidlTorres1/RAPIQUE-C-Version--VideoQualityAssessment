#include "hip/hip_runtime.h"
#include <vector>
#include <numeric>
#include <cmath>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <opencv2/core/ocl.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudafilters.hpp>
#include <execution>
#include <future>

#include "FilterFactory.h"
#include "CUDAFilterFactory.h"



std::vector<double> nakafit(const std::vector<double>& data);
std::pair<double, double> est_GGD_param(const std::vector<double>& vec);
std::tuple<double, double, double> est_AGGD_param(const std::vector<double>& vec);

cv::Mat createManualGaussianKernel() {
    return (cv::Mat_<float>(7, 7) <<
        0.0002, 0.0010, 0.0030, 0.0043, 0.0030, 0.0010, 0.0002,
        0.0010, 0.0062, 0.0187, 0.0270, 0.0187, 0.0062, 0.0010,
        0.0030, 0.0187, 0.0563, 0.0813, 0.0563, 0.0187, 0.0030,
        0.0043, 0.0270, 0.0813, 0.1174, 0.0813, 0.0270, 0.0043,
        0.0030, 0.0187, 0.0563, 0.0813, 0.0563, 0.0187, 0.0030,
        0.0010, 0.0062, 0.0187, 0.0270, 0.0187, 0.0062, 0.0010,
        0.0002, 0.0010, 0.0030, 0.0043, 0.0030, 0.0010, 0.0002);
}

void circularShift(const cv::Mat& src, cv::Mat& dst, cv::Point shift) {
    int shift_x = (shift.x % src.cols + src.cols) % src.cols;
    int shift_y = (shift.y % src.rows + src.rows) % src.rows;

    cv::Mat extended;
    cv::copyMakeBorder(src, extended, 0, shift_y, 0, shift_x, cv::BORDER_WRAP);
    cv::Rect roi(shift_x, shift_y, src.cols, src.rows);

    dst = extended(roi);
}

bool checkKernelEquivalence(const cv::Mat& cppKernel, const cv::Mat& expectedKernel) {
    cv::Mat diff;
    cv::absdiff(cppKernel, expectedKernel, diff);
    double minVal, maxVal;
    cv::Point minLoc, maxLoc;
    cv::minMaxLoc(diff, &minVal, &maxVal, &minLoc, &maxLoc);
    return maxVal < 1e-5;  // tolerance level for float comparisons
}

cv::cuda::GpuMat applyGaussianFilter(const cv::cuda::GpuMat& src, cv::Ptr<cv::cuda::Filter>& filter) {
    cv::cuda::GpuMat dst;
    filter->apply(src, dst);
    return dst;
}


//3128
std::vector<float> rapique_basic_extractor(const cv::Mat& img) {
    CUDAFilterFactory cudaFactory;

    std::vector<float> ftrs;
    ftrs.reserve(18);

    try {
        const int filtlength = 7;
        cv::Mat window = createManualGaussianKernel(); // Ensure this returns cv::Mat

        // Move data to GPU
        cv::cuda::GpuMat img_gpu(img);
        cv::cuda::GpuMat mu_gpu, sigma_sq_gpu, img_sq_gpu, structdis_gpu;

        cv::Ptr<cv::cuda::Filter> filter = cv::cuda::createLinearFilter(img_gpu.type(), -1, window);

        // Gaussian filtering to compute mu
        mu_gpu = applyGaussianFilter(img_gpu, filter);

        // Compute img_sq in-place and then apply Gaussian filtering to compute sigma_sq
        cv::cuda::multiply(img_gpu, img_gpu, img_sq_gpu);
        sigma_sq_gpu = applyGaussianFilter(img_sq_gpu, filter);

        // Correct calculation of sigma on GPU
        cv::cuda::GpuMat mu_squared_gpu;
        cv::cuda::multiply(mu_gpu, mu_gpu, mu_squared_gpu);  // Element-wise squaring of mu
        cv::cuda::subtract(sigma_sq_gpu, mu_squared_gpu, sigma_sq_gpu);
        cv::cuda::max(sigma_sq_gpu, cv::Scalar(0), sigma_sq_gpu);  // Ensure all values are non-negative
        cv::cuda::sqrt(sigma_sq_gpu, sigma_sq_gpu);
        mu_squared_gpu.release();

        // Structural Disimilarity (structdis) calculation on GPU
        cv::cuda::subtract(img_gpu, mu_gpu, structdis_gpu);
        cv::cuda::GpuMat sigma_plus_one_gpu;
        cv::cuda::add(sigma_sq_gpu, cv::Scalar(1), sigma_plus_one_gpu);  // Add 1 to sigma before division
        cv::cuda::divide(structdis_gpu, sigma_plus_one_gpu, structdis_gpu);
        sigma_plus_one_gpu.release();
        structdis_gpu.convertTo(structdis_gpu, CV_32F);

        // Download final results to host for statistical processing
        cv::Mat structdis, sigma;
        structdis_gpu.download(structdis);
        sigma_sq_gpu.download(sigma);
        structdis.convertTo(structdis, CV_32F);
        sigma.convertTo(sigma, CV_32F);

        //stream.waitForCompletion();

        std::vector<double> vec_struct(structdis.begin<float>(), structdis.end<float>());
        auto [gamparam, sigparam] = est_GGD_param(vec_struct);
        ftrs.push_back(gamparam);
        ftrs.push_back(sigparam);

        std::vector<float> sigmaVec(sigma.begin<float>(), sigma.end<float>());
        std::vector<double> sigmaVec_d(sigmaVec.begin(), sigmaVec.end());
        std::vector<double> sigmaParam = nakafit(sigmaVec_d);
        ftrs.insert(ftrs.end(), sigmaParam.begin(), sigmaParam.end());

        const std::vector<std::pair<int, int>> shifts = { {0, 1}, {1, 0}, {1, 1}, {-1, 1} };
        std::vector<cv::Mat> pairs(shifts.size());
        std::transform(std::execution::par, shifts.begin(), shifts.end(), pairs.begin(), [&structdis](const std::pair<int, int>& shift) {
            cv::Mat shifted_structdis;
            circularShift(structdis, shifted_structdis, cv::Point(shift.first, shift.second));
            return structdis.mul(shifted_structdis);
            });

        std::vector<std::future<std::tuple<float, float, float, float>>> futures;
        for (const auto& pair : pairs) {
            futures.push_back(std::async(std::launch::async, [pair]() -> std::tuple<float, float, float, float> {
                std::vector<double> pairVec(pair.begin<float>(), pair.end<float>());
                auto [alpha, leftstd, rightstd] = est_AGGD_param(pairVec); // Assuming est_AGGD_param correctly returns a tuple<float, float, float, float>
                float meanparam = (rightstd - leftstd) * (std::tgamma(2.0f / alpha) / std::tgamma(1.0f / alpha)) *
                    (std::sqrt(std::tgamma(1.0f / alpha)) / std::sqrt(std::tgamma(3.0f / alpha)));
                return std::make_tuple(alpha, meanparam, leftstd, rightstd);
                }));
        }

        for (auto& future : futures) {
            auto [alpha, meanparam, leftstd, rightstd] = future.get();
            ftrs.push_back(alpha);
            ftrs.push_back(meanparam);
            ftrs.push_back(leftstd);
            ftrs.push_back(rightstd);
        }

        cv::Mat log_struct;
        cv::log(cv::abs(structdis) + 0.1, log_struct);

        std::vector<cv::Mat> shifted_structs(shifts.size());
        for (size_t i = 0; i < shifts.size(); ++i) {
            circularShift(log_struct, shifted_structs[i], cv::Point(shifts[i].first, shifts[i].second));
            cv::Mat structdis_diff = log_struct - shifted_structs[i];

            std::vector<double> structdis_diff_vec(structdis_diff.begin<float>(), structdis_diff.end<float>());
            auto [gamparam_diff, sigparam_diff] = est_GGD_param(structdis_diff_vec);
            ftrs.push_back(gamparam_diff);
            ftrs.push_back(sigparam_diff);
        }

        cv::Mat combined_structdis_diff = log_struct + shifted_structs[2] - shifted_structs[0] - shifted_structs[1];
        combined_structdis_diff.convertTo(combined_structdis_diff, CV_32F);
        std::vector<double> combined_diff_vec(combined_structdis_diff.begin<float>(), combined_structdis_diff.end<float>());
        auto [gamparam_combined, sigparam_combined] = est_GGD_param(combined_diff_vec);
        ftrs.push_back(gamparam_combined);
        ftrs.push_back(sigparam_combined);

        // Additional matrix operations using custom filters
        static const cv::Mat win_tmp_1 = (cv::Mat_<float>(3, 3) << 0, 1, 0, -1, 0, -1, 0, 1, 0);
        static const cv::Mat win_tmp_2 = (cv::Mat_<float>(3, 3) << 1, 0, -1, 0, 0, 0, -1, 0, 1);

        cv::Mat structdis_diff_1, structdis_diff_2;
        cv::filter2D(log_struct, structdis_diff_1, CV_32F, win_tmp_1, cv::Point(-1, -1), 0, cv::BORDER_REPLICATE);
        cv::filter2D(log_struct, structdis_diff_2, CV_32F, win_tmp_2, cv::Point(-1, -1), 0, cv::BORDER_REPLICATE);

        std::vector<double> structdis_diff_1_vec(structdis_diff_1.begin<float>(), structdis_diff_1.end<float>());
        std::vector<double> structdis_diff_2_vec(structdis_diff_2.begin<float>(), structdis_diff_2.end<float>());

        auto [gamparam1, sigparam1] = est_GGD_param(structdis_diff_1_vec);
        auto [gamparam2, sigparam2] = est_GGD_param(structdis_diff_2_vec);

        ftrs.push_back(static_cast<float>(gamparam1));
        ftrs.push_back(static_cast<float>(sigparam1));
        ftrs.push_back(static_cast<float>(gamparam2));
        ftrs.push_back(static_cast<float>(sigparam2));

        // Release GPU resources
        img_gpu.release();
        mu_gpu.release();
        sigma_sq_gpu.release();
        img_sq_gpu.release();
        structdis_gpu.release();
    }
    catch (const std::exception& e) {
        std::cerr << "Error in rapique basic extractor computation: " << e.what() << '\n';
    }

    return ftrs;
}


