#include "hip/hip_runtime.h"
#include "YUVReader.h"
#include <fstream>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>  // CUDA Runtime
#include <opencv2/opencv.hpp>  // OpenCV for image processing

// CUDA kernel for resizing images
__global__ void resize_kernel(unsigned char* src, unsigned char* dst, int srcWidth, int srcHeight, int dstWidth, int dstHeight) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < dstWidth && y < dstHeight) {
        float scaleX = (float)srcWidth / dstWidth;
        float scaleY = (float)srcHeight / dstHeight;
        int srcX = (int)(x * scaleX);
        int srcY = (int)(y * scaleY);

        dst[y * dstWidth + x] = src[srcY * srcWidth + srcX];
    }
}

cv::Mat YUVReader::readImage(const std::string& filename, int width, int height, int frameNum) {
    cv::Mat YUV;

    // Declare pointers at the beginning of the function to ensure visibility
    unsigned char* d_Y = nullptr, * d_U = nullptr, * d_V = nullptr;
    unsigned char* d_resizedU = nullptr, * d_resizedV = nullptr;

    try {
        const int frameSize = width * height;
        const int uvFrameSize = frameSize / 4;
        const cv::Size frameDimensions(width, height);

        std::ifstream yuvFile(filename, std::ios::binary);
        if (!yuvFile) {
            std::cerr << "Cannot open file!\n";
            return {};
        }

        yuvFile.seekg(static_cast<long long>(frameSize) * 1.5 * frameNum);

        std::vector<uchar> buffer(frameSize + 2 * uvFrameSize);
        yuvFile.read(reinterpret_cast<char*>(buffer.data()), buffer.size());
        if (yuvFile.gcount() != static_cast<std::streamsize>(buffer.size())) {
            std::cerr << "Error reading the YUV components\n";
            return {};
        }

        hipMalloc(&d_Y, frameSize);
        hipMalloc(&d_U, uvFrameSize);
        hipMalloc(&d_V, uvFrameSize);
        hipMalloc(&d_resizedU, frameSize);
        hipMalloc(&d_resizedV, frameSize);

        hipMemcpy(d_Y, buffer.data(), frameSize, hipMemcpyHostToDevice);
        hipMemcpy(d_U, buffer.data() + frameSize, uvFrameSize, hipMemcpyHostToDevice);
        hipMemcpy(d_V, buffer.data() + frameSize + uvFrameSize, uvFrameSize, hipMemcpyHostToDevice);

        dim3 block(16, 16);
        dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

        resize_kernel << <grid, block >> > (d_U, d_resizedU, width / 2, height / 2, width, height);
        resize_kernel << <grid, block >> > (d_V, d_resizedV, width / 2, height / 2, width, height);
        hipDeviceSynchronize();

        std::vector<cv::Mat> channels(3);
        channels[0] = cv::Mat(frameDimensions, CV_8UC1, buffer.data());  // Y
        channels[1] = cv::Mat(frameDimensions, CV_8UC1);
        channels[2] = cv::Mat(frameDimensions, CV_8UC1);

        hipMemcpy(channels[1].data, d_resizedU, frameSize, hipMemcpyDeviceToHost);
        hipMemcpy(channels[2].data, d_resizedV, frameSize, hipMemcpyDeviceToHost);

        cv::merge(channels, YUV);

    }
    catch (const std::exception& e) {
        std::cerr << "An error occurred reading YUV: " << e.what() << '\n';
    }

    // Clean up CUDA resources in all cases
    hipFree(d_Y);
    hipFree(d_U);
    hipFree(d_V);
    hipFree(d_resizedU);
    hipFree(d_resizedV);

    return YUV;
}
