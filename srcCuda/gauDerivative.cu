#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <opencv2/opencv.hpp>
#include <opencv2/core/ocl.hpp>
#include <future>
#include "Logger.h"
#include "Globals.h"
using namespace std;
using namespace cv;

// CUDA error check macro
#define cudaCheckError() { \
    hipError_t e=hipGetLastError(); \
    if(e!=hipSuccess) { \
        printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
        exit(EXIT_FAILURE); \
    }}

// CUDA Kernel for calculating Gaussian derivatives
__global__ void computeGauDerKernel(double* gauDerX, double* gauDerY,
    int halfLength, double inverseSigmaSquared,
    int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < size && j < size) {
        double iPos = i - halfLength;
        double jPos = j - halfLength;
        double iSquared = iPos * iPos;
        double jSquared = jPos * jPos;
        double commonFactor = exp(-(iSquared + jSquared) * inverseSigmaSquared);
        gauDerX[j * size + i] = iPos * commonFactor;
        gauDerY[j * size + i] = jPos * commonFactor;
    }
}

pair<cv::Mat, cv::Mat> gauDerivative(double sigma) {
    logger.startTimer(video_name_global, "gauDerivative");
    const int halfLength = static_cast<int>(ceil(3 * sigma));
    const int size = 2 * halfLength + 1;
    const double sigmaSquared = 2 * sigma * sigma;
    const double inverseSigmaSquared = 1 / sigmaSquared;
    // Allocate host matrices
    cv::Mat gauDerX(size, size, CV_64F);
    cv::Mat gauDerY(size, size, CV_64F);  // Allocate device memory
    double* d_gauDerX;
    double* d_gauDerY;
    hipMalloc(&d_gauDerX, size * size * sizeof(double));
    hipMalloc(&d_gauDerY, size * size * sizeof(double));
    // Define grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((size + blockDim.x - 1) / blockDim.x,
        (size + blockDim.y - 1) / blockDim.y);
    // Launch kernel
    computeGauDerKernel << <gridDim, blockDim >> > (d_gauDerX, d_gauDerY, halfLength,
        inverseSigmaSquared, size);
    cudaCheckError(); // Check for kernel launch errors
    // Copy result back to host
    hipMemcpy(gauDerX.data, d_gauDerX, size * size * sizeof(double),
        hipMemcpyDeviceToHost);
    hipMemcpy(gauDerY.data, d_gauDerY, size * size * sizeof(double),
        hipMemcpyDeviceToHost);
    cudaCheckError(); // Check for copy errors
    // Free device memory
    hipFree(d_gauDerX);
    hipFree(d_gauDerY);

    logger.stopTimer(video_name_global, "gauDerivative");
    return { gauDerX, gauDerY };
}
