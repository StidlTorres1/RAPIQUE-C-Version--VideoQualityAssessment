#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <numeric>
#include <algorithm>
#include <execution>
#include <iostream>
#include "Logger.h"
#include "Globals.h"

std::pair<double, double> est_GGD_param(const std::vector<double>& vec) {
    logger.startTimer(video_name_global, "est_GGD_param");

    if (vec.empty()) {
        logger.stopTimer(video_name_global, "est_GGD_param");
        return { 0.0, 0.0 };
    }

    // Define gam range and step size
    const double gam_start = 0.1;
    const double gam_end = 6.0;
    const double gam_step = 0.001;
    const size_t gamSize = static_cast<size_t>((gam_end - gam_start) / gam_step + 1);

    std::vector<double> gam(gamSize);
    std::vector<double> r_gam(gamSize);

    // Initialize gam values
    for (size_t i = 0; i < gamSize; ++i) {
        gam[i] = gam_start + i * gam_step;
    }

    // Calculate statistics needed for rho
    double sumAbsX = std::accumulate(vec.begin(), vec.end(), 0.0, [](double acc, double x) { return acc + std::abs(x); });
    double sumX2 = std::accumulate(vec.begin(), vec.end(), 0.0, [](double acc, double x) { return acc + x * x; });
    double sigma_sq = sumX2 / vec.size();
    double E = sumAbsX / vec.size();
    double rho = sigma_sq / (E * E);
    // Compute r_gam using parallel execution
    std::transform(std::execution::par, gam.begin(), gam.end(), r_gam.begin(), [](double gamVal) {
        double tgamma1 = std::tgamma(1.0 / gamVal);
        double tgamma2 = std::tgamma(2.0 / gamVal);
        double tgamma3 = std::tgamma(3.0 / gamVal);
        return (tgamma1 * tgamma3) / (tgamma2 * tgamma2);
        });

    // Find the gamma value that minimizes the difference to rho
    auto it = std::min_element(r_gam.begin(), r_gam.end(), [rho](double a, double b) { return std::abs(a - rho) < std::abs(b - rho); });
    size_t idx = std::distance(r_gam.begin(), it);

    double beta_par = gam[idx];
    double alpha_par = std::sqrt(sigma_sq);

    logger.stopTimer(video_name_global, "est_GGD_param");
    return { beta_par, alpha_par };
}