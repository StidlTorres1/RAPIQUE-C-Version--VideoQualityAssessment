#include "hip/hip_runtime.h"
#include <vector>
#include <numeric>
#include <cmath>
#include <iostream>
#include "Logger.h"
#include "Globals.h"

std::vector<double> nakafit(const std::vector<double>& data) {
    logger.startTimer(video_name_global, "nakafit");
    if (data.empty()) {
        return { 0.0, 0.0 };
    }

    const auto n = data.size();
    const double mean = std::accumulate(data.begin(), data.end(), 0.0) / n;

    const double sq_sum = std::inner_product(data.begin(), data.end(), data.begin(), 0.0,
        std::plus<>(),
        [mean](double a, double b) { return (a - mean) * (b - mean); });

    const double stdev = std::sqrt(sq_sum / (n - 1));
    const double mean_over_stdev_sq = (mean / stdev) * (mean / stdev);
    logger.stopTimer(video_name_global, "nakafit");
    return { mean, mean_over_stdev_sq };
}
