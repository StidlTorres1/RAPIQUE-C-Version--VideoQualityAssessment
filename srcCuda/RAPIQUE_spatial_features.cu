#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudafilters.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/cudawarping.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <omp.h>
#include "Logger.h"
#include "Globals.h"
using namespace std;

#include "CudaOps.h"

// Declaration of utility functions
pair<vector<cv::cuda::GpuMat>, vector<cv::cuda::GpuMat>> gen_DoG(const cv::cuda::GpuMat& img, int kband);
vector<float> rapique_basic_extractor(const cv::Mat& src);
cv::cuda::GpuMat convertRGBToLABCUDA(const cv::cuda::GpuMat& I);

vector<float> RAPIQUE_spatial_features(const cv::Mat& RGB) {
    // Check for empty image and return immediately if true
    if (RGB.empty()) {
        std::cerr << "Received empty input image in RAPIQUE_spatial_features." << std::endl;
        return {};
    }

    logger.startTimer(video_name_global, "RAPIQUE_spatial_features");
    vector<float> feats;
    feats.reserve(680); // Assuming 680 features to be extracted

    try {
        // Validate input image is indeed an RGB image
        if (RGB.channels() != 3) {
            throw invalid_argument("The input should be an RGB image");
        }

        auto cudaOps = std::make_shared<CudaOps>();

        cv::cuda::Stream stream;

        const int kscale = 2;
        const int kband = 4;

        cv::cuda::GpuMat d_RGB(RGB), d_Y;
        cudaOps->cvtColor(d_RGB, d_Y, cv::COLOR_BGR2GRAY);
        d_Y.convertTo(d_Y, CV_32F, 1, 0, stream);

        // Sobel filter operations on the GPU for edge detection
        cv::cuda::GpuMat d_GM_X, d_GM_Y;
        auto sobelFilterX = cudaOps->createSobelFilter(CV_32F, CV_32F, 1, 0, 3);
        auto sobelFilterY = cudaOps->createSobelFilter(CV_32F, CV_32F, 0, 1, 3);
        sobelFilterX->apply(d_Y, d_GM_X);
        sobelFilterY->apply(d_Y, d_GM_Y);

        // Compute magnitude of gradients
        cv::cuda::GpuMat d_GM;
        cudaOps->magnitude(d_GM_X, d_GM_Y, d_GM);
        d_GM_X.release();
        d_GM_Y.release();

        cv::Ptr<cv::cuda::Filter> gaussianFilter = cudaOps->createGaussianFilter(CV_32F, -1, cv::Size(9, 9), 1.5);
        cv::cuda::GpuMat d_LOG;
        gaussianFilter->apply(d_Y, d_LOG, stream);

        stream.waitForCompletion();

        cv::cuda::GpuMat d_Y_float;
        d_Y.convertTo(d_Y_float, CV_32F, 1, 0, stream);
        auto [d_gspace_img, d_ksplit_img] = gen_DoG(d_Y_float, kband);

        cv::cuda::GpuMat d_LAB = convertRGBToLABCUDA(d_RGB);
        d_RGB.release();

        cv::cuda::GpuMat d_channels[3];
        cv::cuda::split(d_LAB, d_channels, stream);
        d_LAB.release();

        cv::cuda::GpuMat d_O1, d_O2;
        cv::cuda::addWeighted(d_channels[0], 0.30, d_channels[1], 0.04, 0, d_O1, -1, stream);
        cv::cuda::addWeighted(d_O1, 1.0, d_channels[2], -0.35, 0, d_O1, -1, stream);

        cv::cuda::addWeighted(d_channels[0], 0.34, d_channels[1], -0.60, 0, d_O2, -1, stream);
        cv::cuda::addWeighted(d_O2, 1.0, d_channels[2], 0.17, 0, d_O2, -1, stream);

        auto computeMagnitude = [](const cv::Mat& src) -> cv::Mat {
            cv::cuda::GpuMat d_src(src), d_Ix, d_Iy;
            cv::Ptr<cv::cuda::Filter> filterX = cv::cuda::createSobelFilter(src.type(), -1, 1, 0, 3);
            cv::Ptr<cv::cuda::Filter> filterY = cv::cuda::createSobelFilter(src.type(), -1, 0, 1, 3);
            filterX->apply(d_src, d_Ix);
            filterY->apply(d_src, d_Iy);
            d_Ix.convertTo(d_Ix, CV_32F);
            d_Iy.convertTo(d_Iy, CV_32F);
            cv::cuda::GpuMat d_magnitude;
            cv::cuda::magnitude(d_Ix, d_Iy, d_magnitude);
            cv::Mat magnitude;
            d_magnitude.download(magnitude);
            return magnitude;
            };

        cv::Mat O1, O2, Y, GM, LOG;
        vector<cv::Mat> channels(3);

        d_O1.download(O1);
        d_O2.download(O2);

        for (int i = 0; i < 3; ++i) {
            d_channels[i].download(channels[i]);
        }

        d_Y.download(Y);
        d_GM.download(GM);
        d_LOG.download(LOG);

        d_O1.release();
        d_O2.release();
        d_GM.release();
        d_LOG.release();
        d_Y.release();

        cv::Mat GMO1 = computeMagnitude(O1);
        cv::Mat GMO2 = computeMagnitude(O2);

        vector<cv::Mat> logChannels(3);
#pragma omp parallel for
        for (int i = 0; i < 3; ++i) {
            channels[i].convertTo(channels[i], CV_32F);
            cv::log(channels[i] + 0.1, logChannels[i]);
        }

        cv::Mat BY = (logChannels[0] - cv::mean(logChannels[0])[0] + logChannels[1] - cv::mean(logChannels[1])[0] - 2 * (logChannels[2] - cv::mean(logChannels[2])[0])) / sqrt(6);
        cv::Mat RG = (logChannels[0] - cv::mean(logChannels[0])[0] - (logChannels[1] - cv::mean(logChannels[1])[0])) / sqrt(2);

        cv::Mat GMBY = computeMagnitude(BY);
        cv::Mat GMRG = computeMagnitude(RG);

        cv::Mat GMA = computeMagnitude(channels[1]);
        cv::Mat GMB = computeMagnitude(channels[2]);

        vector<cv::Mat> ksplit_img(d_ksplit_img.size());

        for (size_t i = 0; i < d_ksplit_img.size(); ++i) {
            d_ksplit_img[i].download(ksplit_img[i]);
        }

        vector<cv::Mat> compositeMat = { Y, GM, LOG };
        if (!ksplit_img.empty()) {
            compositeMat.push_back(ksplit_img[0]);
        }
        compositeMat.insert(compositeMat.end(), { O1, O2, GMO1, GMO2, BY, RG, GMBY, GMRG, channels[1], channels[2], GMA, GMB });

        vector<cv::Mat> scaledMats;
        scaledMats.reserve(compositeMat.size() * kscale);

#pragma omp parallel
        {
            vector<cv::Mat> localScaledMats;
            localScaledMats.reserve(compositeMat.size() * kscale);

#pragma omp for nowait
            for (size_t i = 0; i < compositeMat.size(); ++i) {
                const auto& mat = compositeMat[i];
                for (int scale = 1; scale <= kscale; ++scale) {
                    if (i >= 4 && scale == 1) continue;
                    cv::cuda::GpuMat d_mat(mat), d_y_scale;
                    double scale_factor = pow(2, -(scale - 1));
                    int new_cols = cvRound(mat.cols * scale_factor);
                    int new_rows = cvRound(mat.rows * scale_factor);
                    cudaOps->resize(d_mat, d_y_scale, cv::Size(new_cols, new_rows), 0, 0, cv::INTER_CUBIC);

                    cv::Mat y_scale;
                    d_y_scale.download(y_scale);
                    localScaledMats.push_back(move(y_scale));
                }
            }

#pragma omp critical
            scaledMats.insert(scaledMats.end(), localScaledMats.begin(), localScaledMats.end());
        }

#pragma omp parallel
        {
            vector<float> localFeats;
            localFeats.reserve(680);

#pragma omp for nowait
            for (size_t idx = 0; idx < scaledMats.size(); ++idx) {
                vector<float> chFeats = rapique_basic_extractor(scaledMats[idx]);
                localFeats.insert(localFeats.end(), chFeats.begin(), chFeats.end());
            }

#pragma omp critical
            feats.insert(feats.end(), localFeats.begin(), localFeats.end());
        }
        logger.stopTimer(video_name_global, "RAPIQUE_spatial_features");
        d_Y.release();
        d_GM_X.release();
        d_GM_Y.release();
        d_GM.release();
        d_LOG.release();
        return feats;
    }
    catch (const std::exception& e) {
        std::cerr << "An error occurred during rapique spatial features computation: " << e.what() << '\n';
    }
    return feats;
}

cv::cuda::GpuMat convertRGBToLABCUDA(const cv::cuda::GpuMat& d_I) {
    cv::cuda::GpuMat d_LAB;
    cv::cuda::cvtColor(d_I, d_LAB, cv::COLOR_BGR2Lab, 0);
    cv::cuda::GpuMat d_LAB_float;
    d_LAB.convertTo(d_LAB_float, CV_32F, 1.0, 0.0);
    return d_LAB_float;
}
