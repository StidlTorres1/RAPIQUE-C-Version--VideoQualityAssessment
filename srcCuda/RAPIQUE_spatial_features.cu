#include "hip/hip_runtime.h"
// Include necessary headers for handling vectors, mathematical operations, and OpenCV functionalities
#include <vector>
#include <cmath>
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudafilters.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/cudawarping.hpp>
#include <opencv2/cudaimgproc.hpp> // Include for cuda::cvtColor functionality
#include <omp.h>
#include "Logger.h"
#include "Globals.h"
using namespace std;

// Include the header for the CudaOps class which abstracts CUDA operations
#include "CudaOps.h"

// Declaration of potential utility functions used within the image quality assessment process
pair<vector<cv::cuda::GpuMat>, vector<cv::cuda::GpuMat>> gen_DoG(const cv::cuda::GpuMat& img, int kband);
vector<float> rapique_basic_extractor(const cv::Mat& src);
cv::cuda::GpuMat convertRGBToLABCUDA(const cv::cuda::GpuMat& I);

/**
 * Execute spatial feature extraction on an input RGB image.
 *
 * @param RGB The input RGB image as a cv::Mat.
 * @return A vector of floating-point values representing the extracted spatial features.
 */
vector<float> RAPIQUE_spatial_features(const cv::Mat& RGB) {
    logger.startTimer(video_name_global, "RAPIQUE_spatial_features");
    // Reserve space for feature vector
    vector<float> feats;
    feats.reserve(680); // Assuming 680 features to be extracted

    try{
        // Validate input image is indeed an RGB image
        if (RGB.channels() != 3) {
            throw invalid_argument("The input should be an RGB image");
        }

        // Initialize CUDA operations wrapper class
        auto cudaOps = std::make_shared<CudaOps>();

        cv::cuda::Stream stream;

        // Definition of scales and bands for processing
        const int kscale = 2;
        const int kband = 4;

        // Convert input image to grayscale using GPU
        cv::cuda::GpuMat d_RGB(RGB), d_Y;
        cudaOps->cvtColor(d_RGB, d_Y, cv::COLOR_BGR2GRAY);
        d_Y.convertTo(d_Y, CV_32F, 1, 0, stream);

        // Sobel filter operations on the GPU for edge detection
        cv::cuda::GpuMat d_GM_X, d_GM_Y;
        auto sobelFilterX = cudaOps->createSobelFilter(CV_32F, CV_32F, 1, 0, 3);
        auto sobelFilterY = cudaOps->createSobelFilter(CV_32F, CV_32F, 0, 1, 3);
        sobelFilterX->apply(d_Y, d_GM_X);
        sobelFilterY->apply(d_Y, d_GM_Y);

        // Compute magnitude of gradients
        cv::cuda::GpuMat d_GM;
        cudaOps->magnitude(d_GM_X, d_GM_Y, d_GM);
        d_GM_X.release(); // Release as soon as it's no longer needed
        d_GM_Y.release(); // Release as soon as it's no longer needed


        // Apply Gaussian filter for smoothing
        cv::Ptr<cv::cuda::Filter> gaussianFilter = cudaOps->createGaussianFilter(CV_32F, -1, cv::Size(9, 9), 1.5);
        cv::cuda::GpuMat d_LOG;
        gaussianFilter->apply(d_Y, d_LOG, stream);

        // Wait for all GPU operations to complete
        stream.waitForCompletion();

        // Generate Difference of Gaussians (DoG) for image using the modified gen_DoG
        cv::cuda::GpuMat d_Y_float;
        d_Y.convertTo(d_Y_float, CV_32F, 1, 0, stream);
        auto [d_gspace_img, d_ksplit_img] = gen_DoG(d_Y_float, kband);

        // Convert input image to LAB color space using GPU with the modified convertRGBToLABCUDA
        cv::cuda::GpuMat d_LAB = convertRGBToLABCUDA(d_RGB);
        d_RGB.release();

        // Compute orientation channels O1 and O2 from LAB components directly on the GPU
        cv::cuda::GpuMat d_channels[3];
        cv::cuda::split(d_LAB, d_channels, stream);
        d_LAB.release();

        cv::cuda::GpuMat d_O1, d_O2;
        cv::cuda::addWeighted(d_channels[0], 0.30, d_channels[1], 0.04, 0, d_O1, -1, stream);
        cv::cuda::addWeighted(d_O1, 1.0, d_channels[2], -0.35, 0, d_O1, -1, stream);

        cv::cuda::addWeighted(d_channels[0], 0.34, d_channels[1], -0.60, 0, d_O2, -1, stream);
        cv::cuda::addWeighted(d_O2, 1.0, d_channels[2], 0.17, 0, d_O2, -1, stream);



        // Lambda function to compute magnitude on the GPU
        auto computeMagnitude = [](const cv::Mat& src) -> cv::Mat {
            cv::cuda::GpuMat d_src(src), d_Ix, d_Iy;
            cv::Ptr<cv::cuda::Filter> filterX = cv::cuda::createSobelFilter(src.type(), -1, 1, 0, 3);
            cv::Ptr<cv::cuda::Filter> filterY = cv::cuda::createSobelFilter(src.type(), -1, 0, 1, 3);
            filterX->apply(d_src, d_Ix);
            filterY->apply(d_src, d_Iy);
            d_Ix.convertTo(d_Ix, CV_32F);
            d_Iy.convertTo(d_Iy, CV_32F);
            cv::cuda::GpuMat d_magnitude;
            cv::cuda::magnitude(d_Ix, d_Iy, d_magnitude);
            cv::Mat magnitude;
            d_magnitude.download(magnitude);
            return magnitude;
            };

        cv::Mat O1, O2, Y, GM, LOG;
        vector<cv::Mat> channels(3);

        // Download each GPU matrix to its corresponding CPU matrix
        d_O1.download(O1);
        d_O2.download(O2);

        // Download the channel images from the GPU
        for (int i = 0; i < 3; ++i) {
            d_channels[i].download(channels[i]);
        }

        d_Y.download(Y);
        d_GM.download(GM);
        d_LOG.download(LOG);

        d_O1.release(); // Make sure to release GPU memory
        d_O2.release();
        d_GM.release();
        d_LOG.release();
        d_Y.release(); // Release after its last usage


        // Computing magnitudes for O1, O2, BY, RG using the computeMagnitude lambda
        cv::Mat GMO1 = computeMagnitude(O1);
        cv::Mat GMO2 = computeMagnitude(O2);

        // Parallel conversion of LAB channels to log scale
        vector<cv::Mat> logChannels(3);
#pragma omp parallel for
        for (int i = 0; i < 3; ++i) {
            channels[i].convertTo(channels[i], CV_32F);
            cv::log(channels[i] + 0.1, logChannels[i]);
        }

        // Compute color opponency features BY and RG
        cv::Mat BY = (logChannels[0] - cv::mean(logChannels[0])[0] + logChannels[1] - cv::mean(logChannels[1])[0] - 2 * (logChannels[2] - cv::mean(logChannels[2])[0])) / sqrt(6);
        cv::Mat RG = (logChannels[0] - cv::mean(logChannels[0])[0] - (logChannels[1] - cv::mean(logChannels[1])[0])) / sqrt(2);

        cv::Mat GMBY = computeMagnitude(BY);
        cv::Mat GMRG = computeMagnitude(RG);

        // Additional magnitude computations
        cv::Mat GMA = computeMagnitude(channels[1]);
        cv::Mat GMB = computeMagnitude(channels[2]);

        // Preparing a composite list of matrices for scaling and feature extraction

        vector<cv::Mat> ksplit_img(d_ksplit_img.size());

        // Download each GPU matrix to its corresponding CPU matrix
        for (size_t i = 0; i < d_ksplit_img.size(); ++i) {
            d_ksplit_img[i].download(ksplit_img[i]);
        }

        vector<cv::Mat> compositeMat = { Y, GM, LOG };
        if (!ksplit_img.empty()) {
            compositeMat.push_back(ksplit_img[0]);
        }
        compositeMat.insert(compositeMat.end(), { O1, O2, GMO1, GMO2, BY, RG, GMBY, GMRG, channels[1], channels[2], GMA, GMB });



        vector<cv::Mat> scaledMats;
        scaledMats.reserve(compositeMat.size() * kscale);

        // Parallel scaling of composite matrices to generate scaled versions
#pragma omp parallel
        {
            vector<cv::Mat> localScaledMats;
            localScaledMats.reserve(compositeMat.size() * kscale);

#pragma omp for nowait
            for (size_t i = 0; i < compositeMat.size(); ++i) {
                const auto& mat = compositeMat[i];
                for (int scale = 1; scale <= kscale; ++scale) {
                    if (i >= 4 && scale == 1) continue; // Skip scaling for certain matrices based on their index and scale
                    cv::cuda::GpuMat d_mat(mat), d_y_scale;
                    double scale_factor = pow(2, -(scale - 1));
                    int new_cols = cvRound(mat.cols * scale_factor);
                    int new_rows = cvRound(mat.rows * scale_factor);
                    cudaOps->resize(d_mat, d_y_scale, cv::Size(new_cols, new_rows), 0, 0, cv::INTER_CUBIC);

                    cv::Mat y_scale;
                    d_y_scale.download(y_scale);
                    localScaledMats.push_back(move(y_scale));
                }
            }

#pragma omp critical
            scaledMats.insert(scaledMats.end(), localScaledMats.begin(), localScaledMats.end());
        }

        // Parallel feature extraction from scaled matrices
#pragma omp parallel
        {
            vector<float> localFeats;
            localFeats.reserve(680);

#pragma omp for nowait
            for (size_t idx = 0; idx < scaledMats.size(); ++idx) {
                vector<float> chFeats = rapique_basic_extractor(scaledMats[idx]);
                localFeats.insert(localFeats.end(), chFeats.begin(), chFeats.end());
            }

#pragma omp critical
            feats.insert(feats.end(), localFeats.begin(), localFeats.end());
        }
        logger.stopTimer(video_name_global, "RAPIQUE_spatial_features");
        d_Y.release();
        d_GM_X.release();
        d_GM_Y.release();
        d_GM.release();
        d_LOG.release();
        return feats;
    }
    catch (const std::exception& e) {
        std::cerr << "An error occurred during rapique spatial features computation: " << e.what() << '\n';
    }
}

/**
 * Convert an RGB image to the LAB color space using CUDA acceleration.
 *
 * @param I The input RGB image as a cv::Mat.
 * @return The LAB-converted image as a cv::Mat.
 */




cv::cuda::GpuMat convertRGBToLABCUDA(const cv::cuda::GpuMat& d_I) {
    cv::cuda::GpuMat d_LAB;
    // Apply CUDA accelerated color conversion
    cv::cuda::cvtColor(d_I, d_LAB, cv::COLOR_BGR2Lab, 0);
    // Convert the LAB image to floating point format on the GPU
    cv::cuda::GpuMat d_LAB_float;
    d_LAB.convertTo(d_LAB_float, CV_32F, 1.0, 0.0);
    return d_LAB_float;
}


