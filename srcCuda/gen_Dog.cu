#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <opencv2/opencv.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudafilters.hpp>
#include "Logger.h"
#include "Globals.h"

using namespace std;

pair<vector<cv::cuda::GpuMat>, vector<cv::cuda::GpuMat>> gen_DoG(const cv::cuda::GpuMat& d_img, int kband) {
    if (d_img.empty()) {
        throw invalid_argument("Input image is empty.");
    }
    if (kband <= 0) {
        throw invalid_argument("kband must be positive.");
    }

    logger.startTimer(video_name_global, "gen_DoG");
    constexpr double kval = 1.6;
    vector<cv::cuda::GpuMat> gspace_img(kband);
    vector<cv::cuda::GpuMat> ksplit_img(kband);
    vector<double> sigmas(kband);
    vector<int> wsizes(kband);

    gspace_img[0] = d_img.clone(); // Store the original image for the output

    try {
        // Pre-calculate sigmas and window sizes
        for (int band = 1; band < kband; ++band) {
            sigmas[band] = pow(kval, band - 2);
            int ws = static_cast<int>(ceil(2 * (3 * sigmas[band] + 1)));
            wsizes[band] = ws + (ws % 2 == 0 ? 1 : 0);
        }

        // Parallel Gaussian Blur using CUDA
        for (int band = 1; band < kband; ++band) {
            cv::cuda::GpuMat d_blurred;
            auto filter = cv::cuda::createGaussianFilter(d_img.type(), -1, cv::Size(wsizes[band], wsizes[band]), sigmas[band], sigmas[band], cv::BORDER_REPLICATE);
            filter->apply(d_img, d_blurred);
            gspace_img[band] = d_blurred;
        }

        // Parallel Subtraction using CUDA
        for (int band = 0; band < kband - 1; ++band) {
            cv::cuda::GpuMat d_result;
            cv::cuda::subtract(gspace_img[band], gspace_img[band + 1], d_result, cv::noArray(), -1);
            ksplit_img[band] = d_result;
        }
        ksplit_img[kband - 1] = gspace_img[kband - 1].clone();

        logger.stopTimer(video_name_global, "gen_DoG");
        return { gspace_img, ksplit_img };
    }
    catch (const std::exception& e) {
        std::cerr << "An error occurred during DoG computation: " << e.what() << '\n';
        throw;
    }
}