#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <algorithm>
#include <numeric>
#include <limits>
#include <execution>
#include <future>
#include <iostream>
#include "Logger.h"
#include "Globals.h"


std::pair<float, float> calculateStdDev(const std::vector<double>& vec) {
    float sumLeft = 0.0, sumRight = 0.0;
    int countLeft = 0, countRight = 0;

    for (float val : vec) {
        if (val < 0) {
            sumLeft += std::abs(val);
            ++countLeft;
        }
        else if (val > 0) {
            sumRight += std::abs(val);
            ++countRight;
        }
    }

    float leftMean = countLeft == 0 ? 0.0 : std::sqrt(sumLeft / countLeft);
    float rightMean = countRight == 0 ? 0.0 : std::sqrt(sumRight / countRight);

    return { leftMean, rightMean };
}

std::vector<double> generateGam() {
    std::vector<double> gam;
    for (double g = 0.2; g <= 10.0; g += 0.001) {
        gam.push_back(g);
    }
    return gam;
}


std::tuple<double, double, double> est_AGGD_param(const std::vector<double>& vec) {
    std::vector<double> gam(9951);  // Adjust the size to match MATLAB's 0.2:0.001:10 range
    std::iota(gam.begin(), gam.end(), 200);  // Start from 0.2
    std::transform(gam.begin(), gam.end(), gam.begin(), [](double x) { return x / 1000.0; });

    double sumAbs = std::accumulate(vec.begin(), vec.end(), 0.0,
        [](double acc, double val) { return acc + std::abs(val); });
    double meanAbs = sumAbs / vec.size();

    auto [leftstd, rightstd] = calculateStdDev(vec);

    double gammahat = leftstd / rightstd;
    double rhat = std::pow(meanAbs, 2) / std::accumulate(vec.begin(), vec.end(), 0.0, [](double acc, double val) { return acc + val * val; }) / vec.size();
    double rhatnorm = (rhat * (std::pow(gammahat, 3) + 1) * (gammahat + 1)) / std::pow((std::pow(gammahat, 2) + 1), 2);

    double minDiff = std::numeric_limits<double>::max();
    double alpha = 0.0;

    auto result = std::transform_reduce(
        std::execution::par,
        gam.begin(),
        gam.end(),
        std::make_pair(std::numeric_limits<double>::max(), 0.0),
        [](const std::pair<double, double>& a, const std::pair<double, double>& b) {
            return (a.first < b.first) ? a : b;
        },
        [&rhatnorm](double x) {
            double r_gam_val = std::pow(std::tgamma(2.0 / x), 2) / (std::tgamma(1.0 / x) * std::tgamma(3.0 / x));
            double diff = std::pow(r_gam_val - rhatnorm, 2);
            return std::make_pair(diff, x);
        }
    );

    return { result.second, leftstd, rightstd };
}